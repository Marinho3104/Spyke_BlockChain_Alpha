#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void mm() {
    while(1);
}

__host__ void test(void*) {
    printf("Host function\n");
}

int main() {
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipStream_t stream_1;
    hipStreamCreate(&stream_1);

    hipEvent_t event;
    hipEventCreate(&event);


    mm<<<1,1, 0, stream_1>>>();
    hipLaunchHostFunc(stream, test, 0);

    hipEventRecord(event, stream);
    hipStreamWaitEvent(stream, event, 0);

    hipStreamSynchronize(stream_1);

    hipEventDestroy(event);
    hipStreamDestroy(stream);

    return 0;
}

/*
__global__ void handle_share_protocol_object( cudaSemaphore_t* _sem ) {
    int _global_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    printf("Waiting for data ... ");

    cudaSemaphoreWait(semaphore);

    

}

// Stream for share protocol
cudaStream_t share_protocol_handle_object_cuda_stream;

extern "C" {

    // Initialize the stream variable and functions to block threads
    void initialize_cuda_share_protocol() {

    }

}


/*
cudaSemaphore_t semaphore;
    cudaSemaphoreCreate(&semaphore, 1);


*/

