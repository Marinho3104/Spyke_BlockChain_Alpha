#include "hip/hip_runtime.h"
#include <cuda/semaphore>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Defines
#define P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_BLOCKS 1
#define P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS 1024

#define P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS_SEMAPHORES_SIZE \
    ( sizeof(cuda::std::binary_semaphore) * P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS * P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_BLOCKS )


// Stream to add data into thread blocks
hipStream_t stream_add_data;

// Stream to execute all thread blocks
hipStream_t stream_thread_blocks;

// Semaphore that indicates when data is available
cuda::std::binary_semaphore* _sem_data_available;

// Semaphore that indicates if a thread block is working ( 0: Means is working 1: Means is not working)
cuda::std::binary_semaphore* _sem_is_working;


// Function to compute all confirmations about a transactions
__global__ void compute_transaction_confirmations( cuda::std::binary_semaphore* _sem_data_available, cuda::std::binary_semaphore* _sem_is_working ) {

    int _global_id = 
        blockIdx.x * blockDim.x + threadIdx.x;

    while( 1 ) {

        _sem_data_available[ _global_id ].acquire();

        printf("Data received %d\n", _global_id);

        clock_t start = clock64(); // Record the start time
        clock_t end = start + 10 * 1000000000ULL; // Calculate the end time (5 seconds from start)
        while (clock64() < end) {
            // Do nothing, just wait
        }

        printf("Done");

        _sem_is_working[ _global_id ].release();

    }

}

void add_transaction_data_into_stream( void* __args ) {

    for ( int _ = 0; _ < P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS * P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_BLOCKS; _++ ) 

        if ( _sem_is_working[ _ ].try_acquire() ) { _sem_data_available[ _ ].release(); break; }

}


extern "C++" {

    // Initialize all needed variables
    void initialize_share_protocol_cuda() {

        // Creates the streams
        hipStreamCreate(&stream_add_data);
        hipStreamCreate(&stream_thread_blocks);

        // Heap allocation for semaphores 
        hipMallocManaged( &_sem_data_available, P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS_SEMAPHORES_SIZE );
        hipMallocManaged( &_sem_is_working, P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS_SEMAPHORES_SIZE );

        // Inicialize semaphores 
        for (int _ = 0; _ < P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS * P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_BLOCKS; _++) {

            new(_sem_data_available + _) cuda::std::binary_semaphore(0); new(_sem_is_working + _) cuda::std::binary_semaphore(1);

        }

        compute_transaction_confirmations<<<P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_BLOCKS, P2P_PROTOCOLS_SHARE_PROTOCOL_CUDA_THREAD_BLOCKS, 0, stream_thread_blocks>>>( _sem_data_available, _sem_is_working );

    }

    // Adds a new host function into the stream to be executed
    void add_transaction_data_share_protocol_cuda(unsigned char __object_count, void* __data, unsigned long long __data_size) {
    
        hipLaunchHostFunc(stream_add_data, add_transaction_data_into_stream, 0);

        hipEvent_t event;
        hipEventCreate(&event);

        hipEventRecord(event, stream_add_data);

        hipError_t err = hipEventSynchronize(event);
        if (err != hipSuccess) {
            printf("Error synchronizing event: %s\n", hipGetErrorString(err));
            return;
        }
    
    }

}

